#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>

//#include <string>
//#include <fstream>
//#include <iterator>
#include <hip/hip_runtime.h>

#include <stdlib.h>

//#include "EquelleRuntimeCUDA.hpp"
#include "EquelleRuntimeCUDA_cuda.hpp"

// Implementation of the class CollOfScalar

CollOfScalar::CollOfScalar()
{
    //values = 0;
    size = 0;
    dev_values = 0;
    //dev_vec = thrust::device_vector<double>(0);

}

CollOfScalar::CollOfScalar(int size) {
    // dev_vec.reserve(size);
    this->size = size;
    //values = (double*)malloc(size*sizeof(double));
    //dev_vec = thrust::device_vector<double>(size);
    hipError_t status = hipMalloc( (void**)&dev_values, size*sizeof(double));
    if ( status != hipSuccess ) {
	std::cout << "Error allocating dev_values in CollOfScalar(int)\n";
	exit(0);
    }
}


// Copy constructor
CollOfScalar::CollOfScalar(const CollOfScalar& coll) {
    std::cout << "Copy constructor!\n";
    size = coll.size;
    //values = 0;
    dev_values = 0;
    //if (coll.values != 0) {
    //	values = (double*)malloc(size*sizeof(double));
    //	for ( int i = 0; i < size; i++) {
    //	    values[i] = coll.values[i];
    //	}
    //}
    if (coll.dev_values != 0) {
	hipError_t status = hipMalloc( (void**)&dev_values, size*sizeof(double));
	if ( status != hipSuccess ) {
	    std::cout << "Error allocating dev_values in CollOfScalar(CollOfScalar)\n";
	    exit(0);
	}
	status = hipMemcpy(dev_values, coll.dev_values, size*sizeof(double),
			    hipMemcpyDeviceToDevice);
	if ( status != hipSuccess ){
	    std::cout << "Error copying dev_values in copy constructor\n";
	    exit(0);
	}
    }    
}


// Destructor:
CollOfScalar::~CollOfScalar() {
    if ( size > 0 ) {
	size = 0;
    }
    //if (values != 0) {
    //	std::cout << "Freeing values\n";
    //	free(values);
    //	//values = 0;
    //}
    if (dev_values != 0) {
	hipError_t status = hipFree(dev_values);
	if (status != hipSuccess) {
	    std::cout << "Error cuda-freeing in destructor of CollOfScalar\n";
	    std::cout << "\tError code: " << hipGetErrorString(status) << std::endl;
	    exit(0);
	}
	//dev_values = 0;
    }
}

double* CollOfScalar::getDevValues() const {
    return dev_values;
}


// Assumes that values are already allocated on host
void CollOfScalar::copyToHost(double* values) const
{
    std::cout << "copyToHost() - dev_values = " << dev_values << std::endl;
    
    hipError_t hipError_t = hipMemcpy( values, dev_values, size*sizeof(double),
					hipMemcpyDeviceToHost);
    if (hipError_t != hipSuccess) {
	std::cout << "Error copying to host in output. \n\tError code = ";
	std::cout << hipGetErrorString(hipError_t) << "\n";
	exit(0);
    }
}


void CollOfScalar::setValuesFromFile(std::istream_iterator<double> begin,
				     std::istream_iterator<double> end)
{
    thrust::host_vector<double> host_vec(begin, end);

    double* values = (double*)malloc(size*sizeof(double));
    for(int i = 0; i < host_vec.size(); i++) {
	values[i] = host_vec[i];
    }
    //dev_vec = host_vec;
    hipError_t cudaStatus = hipMemcpy( dev_values, values, size*sizeof(double),
					 hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
	std::cout << "Error in hipMemcpy to dev from file.\n";
	exit(0);
    }
    free(values);
}

void CollOfScalar::setValuesUniform(double val)
{
    // Can not use hipMemset as it sets float values on a given
    // number of bytes.
    double* host = (double*)malloc(size*sizeof(double));
    for (int i = 0; i < size; ++i) {
	host[i] = val;
    }
    
    hipError_t status = hipMemcpy(dev_values, host, size*sizeof(double),
				    hipMemcpyHostToDevice);
    if ( status != hipSuccess ) {
	std::cout << "Error in uniform value initialization\n";
	std::cout << "\tError code: " << hipGetErrorString(status) << std::endl;
	exit(0);
    }
    free(host);
}

int CollOfScalar::getSize() const
{
    //return dev_vec.size();
    return size;
}
